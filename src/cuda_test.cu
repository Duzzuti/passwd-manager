#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

//#include "cuda_test.h"

// CUDA kernel for matrix multiplication
__global__ void matrixMultiply(int* A, int* B, int* C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        int sum = 0;
        for (int k = 0; k < N; ++k) {
            sum += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}
int main() {
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);

    if (deviceCount == 0) {
        std::cerr << "No GPUs detected." << std::endl;
        return 1;
    }

    for (int i = 0; i < deviceCount; ++i) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, i);

        std::cout << "GPU Device " << i << ": " << deviceProp.name << std::endl;
        std::cout << "  Compute Capability: " << deviceProp.major << "." << deviceProp.minor << std::endl;
        std::cout << "  Total Global Memory: " << deviceProp.totalGlobalMem << " bytes" << std::endl;
        std::cout << "  Multiprocessors: " << deviceProp.multiProcessorCount << std::endl;
        std::cout << "  Clock Rate: " << deviceProp.clockRate / 1000 << " MHz" << std::endl;
        std::cout << "  Memory Clock Rate: " << deviceProp.memoryClockRate / 1000 << " MHz" << std::endl;
        std::cout << "  Memory Bus Width: " << deviceProp.memoryBusWidth << " bits" << std::endl;
    }

    return 0;
}