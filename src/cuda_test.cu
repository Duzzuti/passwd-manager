#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

//#include "cuda_test.h"

// CUDA kernel
__global__ void cudaAdd(unsigned char* a, unsigned char* b, unsigned char* c, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        c[idx] = a[idx] + b[idx] + (a[idx] + b[idx])*(a[idx] + b[idx]);
    }
}
int main() {
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);

    if (deviceCount == 0) {
        std::cerr << "No GPUs detected." << std::endl;
        return 1;
    }

    for (int i = 0; i < deviceCount; ++i) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, i);

        std::cout << "GPU Device " << i << ": " << deviceProp.name << std::endl;
        std::cout << "  Compute Capability: " << deviceProp.major << "." << deviceProp.minor << std::endl;
        std::cout << "  Total Global Memory: " << deviceProp.totalGlobalMem << " bytes" << std::endl;
        std::cout << "  Multiprocessors: " << deviceProp.multiProcessorCount << std::endl;
        std::cout << "  Clock Rate: " << deviceProp.clockRate / 1000 << " MHz" << std::endl;
        std::cout << "  Memory Clock Rate: " << deviceProp.memoryClockRate / 1000 << " MHz" << std::endl;
        std::cout << "  Memory Bus Width: " << deviceProp.memoryBusWidth << " bits" << std::endl;
    }
    const int size = 1024*1024*1024;    // 10 MB
    unsigned char* a = new unsigned char[size];
    unsigned char* b = new unsigned char[size];
    unsigned char* d_a, *d_b, *d_c;

    // Allocate GPU memory
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);
    // Copy data from host to GPU
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 32;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    
    for(int i = 0; i < 10; i++){
    hipMemcpyAsync(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpyAsync(d_b, b, size, hipMemcpyHostToDevice);
    cudaAdd<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, size);

    std::cout << "Matrix multiplication completed." << std::endl;
    }
    // Free GPU memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Do something with the result in 'c'
    delete[] a;
    delete[] b;

    return 0;
}